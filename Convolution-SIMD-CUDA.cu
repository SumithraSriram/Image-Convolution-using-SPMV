#include "hip/hip_runtime.h"

#define WIN32_LEAN_AND_MEAN
#include <windows.h>
#include <tchar.h>
#include <stdio.h>
#include <string.h>
#include<conio.h>
#include<stdlib.h>
#include<tmmintrin.h>
#include "hip/hip_runtime.h"
#include ""
#include<hip/hip_runtime.h>

#pragma once
#pragma warning(disable : 4035)

inline unsigned __int64 GetCycleCount(void)
{
	_asm    _emit 0x0F
	_asm    _emit 0x31
}

class MyTimer
{
	unsigned __int64  m_startcycle;

public:

	unsigned __int64  m_overhead;

	MyTimer(void)
	{
		m_overhead = 0;
		Start();
		m_overhead = Stop();
	}

	void Start(void)
	{
		m_startcycle = GetCycleCount();
	}

	unsigned __int64 Stop(void)
	{
		return GetCycleCount() - m_startcycle - m_overhead;
	}
};


__global__ void csrSpmv(int *x,int * y,int * ptr,int * indices,int * data,int num_rows)
{
	int row = blockDim.x * blockIdx.x + threadIdx.x ;
	if (row < num_rows)
	{
		int dot = 0;
		for (int j = ptr[row]; j < ptr[row+1]; j++) 
			dot += data[j] * x[indices[j]];

		y[row] = dot;
	}
}

int main()
{
    int k;
    int h;
    int w;
    int * hostInputImageData;
    int * hostOutputImageData;
    int * hostptr;
	int * hostcol;
	int * hostdata;
    int * deviceInputImageData;
    int * deviceOutputImageData;
    int * deviceMaskData;
	int * deviceptr;
	int * devicecol;
	int * devicedata;

    
  FILE *fp, *f1, *f2; 
  int i,j,nnz;
  
  printf("Enter the height of the image: ");
  scanf("%d",&h);
  printf("Enter the width of the image: ");
  scanf("%d",&w);
  
  hostInputImageData = (int *) malloc(w*h*sizeof(int));
  hostOutputImageData = (int *) malloc(w*h*sizeof(int));
  hostptr = (int *) malloc((w*h+1)*sizeof(int));
  
  fp = fopen("Cameraman160.txt", "r");
  
  for(i=0;i<h;i++)
  {
	  for(j=0;j<w;j++)
	  {
		fscanf(fp, "%d ", (hostInputImageData+(w*i)+j));
	  }
  }
  
  fclose(fp);
  
  f1 = fopen("Kernel160.txt", "r");
  
  fscanf(f1,"%d\n",&nnz);

  hostcol = (int *) malloc(nnz*sizeof(int));
  hostdata = (int *) malloc(nnz*sizeof(int));

  printf("nnz=%d\n", nnz);
  for(i=0;i<nnz;i++)
  {
		fscanf(fp, "%d %d\n", (hostcol+i),(hostdata+i));
  }
  for(i=0;i<=(h*w);i++)
  {
		fscanf(fp, "%d ", (hostptr+i));
  }
    
  fclose(f1);

    hipMalloc((void **) &deviceInputImageData, w * h * sizeof(int));
    hipMalloc((void **) &deviceOutputImageData, w * h * sizeof(int));
    hipMalloc((void **) &deviceptr, (w*h +1) * sizeof(int));
	hipMalloc((void **) &devicecol, nnz * sizeof(int));
	hipMalloc((void **) &devicedata, nnz * sizeof(int));

	hipMemcpy(deviceInputImageData, hostInputImageData, w * h * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceptr, hostptr, (w*h +1) * sizeof(int) ,hipMemcpyHostToDevice);
	hipMemcpy(devicecol, hostcol, nnz * sizeof(int) ,hipMemcpyHostToDevice);
	hipMemcpy(devicedata, hostdata, nnz * sizeof(int) ,hipMemcpyHostToDevice);
    
	dim3 block(32,1,1);
	dim3 grid( (w*h)/32 , 1 , 1);

	MyTimer timer;

	timer.Start();                // take initial reading
	Sleep(1000);                // wait a second 
	unsigned cpuspeed10 = (unsigned)(timer.Stop() / 100000);
	
	timer.Start();
	
	csrSpmv<<<grid, block>>>(deviceInputImageData, deviceOutputImageData, deviceptr, devicecol, devicedata, (w*h));	

	unsigned cycle = (unsigned)timer.Stop();
	unsigned speed = (unsigned)(cycle / 100000);

	printf("\n\nTime = %d\n", cycle * 10000 / cpuspeed10);
    
    hipMemcpy(hostOutputImageData,deviceOutputImageData, w * h * sizeof(int),hipMemcpyDeviceToHost);
    

   	f2 = fopen("Output.txt", "w");
  
  for(i=0;i<h;i++)
  {
	  for(j=0;j<w;j++)
		fprintf(f2, "%d ", *(hostOutputImageData+(w*i)+j));
	  fprintf(f2,"\n");
  }
  fclose(f2);

  hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceptr);
	hipFree(devicecol);
	hipFree(devicedata);

    getch();
    return 0;
}
