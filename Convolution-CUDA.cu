#include "hip/hip_runtime.h"

#define WIN32_LEAN_AND_MEAN
#include <windows.h>
#include <tchar.h>
#include <stdio.h>
#include <string.h>
#include<conio.h>
#include<stdlib.h>
#include<tmmintrin.h>
#include "hip/hip_runtime.h"
#include ""
#include<hip/hip_runtime.h>

#define Mask_width  3
#define Mask_radius Mask_width/2
#define O_Tile_Width 8
#define Block_Width (O_Tile_Width + Mask_width -1)

#pragma once
#pragma warning(disable : 4035)

inline unsigned __int64 GetCycleCount(void)
{
	_asm    _emit 0x0F
	_asm    _emit 0x31
}

class MyTimer
{
	unsigned __int64  m_startcycle;

public:

	unsigned __int64  m_overhead;

	MyTimer(void)
	{
		m_overhead = 0;
		Start();
		m_overhead = Stop();
	}

	void Start(void)
	{
		m_startcycle = GetCycleCount();
	}

	unsigned __int64 Stop(void)
	{
		return GetCycleCount() - m_startcycle - m_overhead;
	}
};


__global__ void convolution(int *N, const int * __restrict__ M, int *O, int width, int height)
{
	
	__shared__ float ns[Block_Width][Block_Width];
	
	
	int tx= threadIdx.x;
	int ty = threadIdx.y;
	
	int row_o=blockIdx.y*O_Tile_Width +ty;
	int col_o=blockIdx.x*O_Tile_Width +tx;
	
	int row_i= row_o - Mask_radius;
	int col_i= col_o - Mask_radius;
	int out;
	
	if( (row_i >=0) &&( row_i<height) && (col_i >=0) && (col_i < width))
	{
		ns[ty][tx] = N[(row_i * width + col_i)]; 
	}
	else
	{
		ns[ty][tx] = 0.0f;
	}
	
	__syncthreads();
	
	if( ty< O_Tile_Width && tx< O_Tile_Width )
	{
		out=0.0f;
		for(int i=0;i<Mask_width;i++)
		{
			for(int j=0;j<Mask_width;j++)
			{
				out += M[i * Mask_width + j] * ns[i + ty][j+tx];
			}
		}
	}
	
	__syncthreads();
	
	if((row_o < height) && (col_o < width) &&(tx < O_Tile_Width) && (ty < O_Tile_Width))
	{
		O[(row_o * width + col_o)] = out;
	}
		
}				

int main()
{
    int k;
    int h;
    int w;
    int * hostInputImageData;
    int * hostOutputImageData;
    int * hostMaskData;
    int * deviceInputImageData;
    int * deviceOutputImageData;
    int * deviceMaskData;

    
  FILE *fp, *f1, *f2; 
  int i,j;
  
  printf("Enter the height of the image: ");
  scanf("%d",&h);
  printf("Enter the width of the image: ");
  scanf("%d",&w);
  printf("Enter the size of the kernel: ");
  scanf("%d",&k);

  hostInputImageData = (int *) malloc(w*h*sizeof(int));
  hostOutputImageData = (int *) malloc(w*h*sizeof(int));
  hostMaskData = (int *) malloc(k*k*sizeof(int));

  fp = fopen("Baboon512.txt", "r");
  
  for(i=0;i<h;i++)
  {
	  for(j=0;j<w;j++)
	  {
		fscanf(fp, "%d ", (hostInputImageData+(w*i)+j));
		//printf("%d ", *(hostInputImageData+(w*i)+j));
	  }
	  //printf("\n");
  }
  
  fclose(fp);
  
  f1 = fopen("kernel.txt", "r");
  
  for(i=0;i<k;i++)
  {
	  for(j=0;j<k;j++)
	  {
		fscanf(f1, "%d ", (hostMaskData+(k*i)+j));
		//printf("%d ", *(hostMaskData+(k*i)+j));
	  }
	  //printf("\n");
  }
  fclose(f1);

    hipMalloc((void **) &deviceInputImageData, w * h * sizeof(int));
    hipMalloc((void **) &deviceOutputImageData, w * h * sizeof(int));
    hipMalloc((void **) &deviceMaskData, k * k * sizeof(int));

	hipMemcpy(deviceInputImageData, hostInputImageData, w * h * sizeof(int),hipMemcpyHostToDevice);
    hipMemcpy(deviceMaskData, hostMaskData, k * k * sizeof(int),hipMemcpyHostToDevice);
    
	dim3 block(Block_Width, Block_Width,1);
	dim3 grid( (w-1)/O_Tile_Width +1 , (h-1)/O_Tile_Width +1 , 1);

	MyTimer timer;

	timer.Start();                // take initial reading
	Sleep(1000);                // wait a second 
	unsigned cpuspeed10 = (unsigned)(timer.Stop() / 100000);
	
	timer.Start();
	
	convolution<<<grid, block>>>(deviceInputImageData, deviceMaskData, deviceOutputImageData, w, h);	

	unsigned cycle = (unsigned)timer.Stop();
	unsigned speed = (unsigned)(cycle / 100000);

	printf("\n\nTime = %d\n", cycle * 10000 / cpuspeed10);
    
    hipMemcpy(hostOutputImageData,deviceOutputImageData, w * h * sizeof(int),hipMemcpyDeviceToHost);
    

   	f2 = fopen("OutputB512.txt", "w");
  
  for(i=0;i<h;i++)
  {
	  for(j=0;j<w;j++)
		fprintf(f2, "%d ", *(hostOutputImageData+(w*i)+j));
	  fprintf(f2,"\n");
  }
  fclose(f2);

  hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
    hipFree(deviceMaskData);

    getch();
    return 0;
}
